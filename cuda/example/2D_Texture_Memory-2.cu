#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
//#include "handle.cu"
#include "hip/hip_runtime.h"
#include "common/book.h"

using namespace std;

texture<float,2,hipReadModeElementType> tex_w;

__global__ void kernel(int imax, float (*f)[3])
{
  int i = threadIdx.x;
  int j = threadIdx.y;
  // width = 3, height = imax
  // but we have imax threads in x, 3 in y
  // therefore height corresponds to x threads (i)
  // and width corresponds to y threads (j)
  if(i<imax)
    {
      // linear filtering looks between indices
      // f[i][j] = tex2D(tex_w, j+0.5f, i+0.5f);
      f[i][j] = tex2D(tex_w, j, i);
    }
}

void print_to_stdio(int imax, float (*w)[3])
{
  for (int i=0; i<imax; i++)
    {
      printf("%2d  %3.3f  %3.3f  %3.3f\n",i, w[i][0], w[i][1], w[i][2]);
    }
  printf("\n");
}

int main(void)
{
  int imax = 8;
  float (*w)[3];
  float (*d_f)[3], *d_w;
  dim3 grid(imax,3);

  w = (float (*)[3])malloc(imax*3*sizeof(float));

  for(int i=0; i<imax; i++)
    {
      for(int j=0; j<3; j++)
        {
          w[i][j] = i + 0.01f*j;
        }
    }

  print_to_stdio(imax, w);

  size_t pitch;
  HANDLE_ERROR( hipMallocPitch((void**)&d_w, &pitch, 3*sizeof(float), imax) );

  HANDLE_ERROR( hipMemcpy2D(d_w,             // device destination
                             pitch,           // device pitch (calculated above)
                             w,               // src on host
                             3*sizeof(float), // pitch on src (no padding so just width of row)
                             3*sizeof(float), // width of data in bytes
                             imax,            // height of data
                             hipMemcpyHostToDevice) );

  HANDLE_ERROR( hipBindTexture2D(NULL, tex_w, d_w, tex_w.channelDesc, 3, imax, pitch) );
  /*
  tex_w.normalized = false;  // don't use normalized values
  tex_w.filterMode = hipFilterModeLinear;
  tex_w.addressMode[0] = hipAddressModeClamp; // don't wrap around indices
  tex_w.addressMode[1] = hipAddressModeClamp;
  */

  // d_f will have result array
  hipMalloc( &d_f, 3*imax*sizeof(float) );

  // just use threads for simplicity
  kernel<<<1,grid>>>(imax, d_f);

  hipMemcpy(w, d_f, 3*imax*sizeof(float), hipMemcpyDeviceToHost);

  hipUnbindTexture(tex_w);
  hipFree(d_w);
  hipFree(d_f);

  print_to_stdio(imax, w);

  free(w);
  return 0;
}

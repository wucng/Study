#include <iostream>
#include <hip/hip_runtime.h>

#define mycout cout<<"["<<__FILE__<<":"<<__LINE__<<"] "
#define CHECK(res) if(res!=hipSuccess){exit(-1);}
#define rows 5
#define cols 3

using namespace std;
typedef float FLOAT;

// __global__ void vec_add(FLOAT **a,const int rows,const int cols)
__global__ void vec_add(FLOAT **a)
{
    __shared__ float A[rows][cols];
    int x=threadIdx.x;
    int y=threadIdx.y;
    if(x>=cols || y>=rows) return;
    A[y][x]=a[y][x]+2;
    __syncthreads();

    a[y][x]=A[y][x];
    // a[y][x]+=2;
}

int main()
{
    mycout<<"虚拟统一内存使用(CPU与GPU都能访问) 并使用共享内存\n"<<
    "使用二维数组(二维数组其实可以展开为一维数组处理)"<<endl;

    // int rows=5,cols=3;
    FLOAT **a=nullptr;
    // 分配内存
    // a=(FLOAT**)malloc(rows*sizeof(FLOAT*));
    CHECK(hipMallocManaged((void**)&a,rows*sizeof(FLOAT*)));

    for(int i=0;i<rows;++i)
    {
        // a[i]=(FLOAT *)malloc(cols*sizeof(FLOAT));
        CHECK(hipMallocManaged((void**)&a[i],cols*sizeof(FLOAT)));
    }

    // 赋值
    for(int i=0;i<rows;++i)
    {
        for(int j=0;j<cols;++j)
        {
            a[i][j]=j+i*cols;
        }
    }

    // 启动核函数
    dim3 threads(32,32);
    // vec_add<<<1,threads>>>(a,rows,cols);
    vec_add<<<1,threads>>>(a);

    hipDeviceSynchronize(); //等待GPU执行完成， 有多种方式

    // 打印
    for(int i=0;i<rows;++i)
    {
        for(int j=0;j<cols;++j)
        {
            cout<<a[i][j]<<" ";
        }
        cout<<endl;
    }

    // free
    for(int i=0;i<rows;++i)
    {
        if(a[i]!=NULL)
            // free(a[i]);
            CHECK(hipFree(a[i]));
    }
    if(a!=NULL)
        // free(a);
        CHECK(hipFree(a));

    return 0;
}

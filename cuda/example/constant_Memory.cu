#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <ctime>
#include "common/book.h"

#define mycout cout<<"["<<__FILE__<<":"<<__LINE__<<"] "

/* 全局线程id get thread id: 1D block and 2D grid  <<<(32,32),32>>>*/
#define get_tid() (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x)  // 2D grid,1D block
// #define get_tid() (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x+threadIdx.y*blockDim.x)  // 2D grid,2D block

/* get block id: 2D grid */
#define get_bid() (blockIdx.x + blockIdx.y * gridDim.x)

/* 每个block的线程id*/
// #define get_tid_per_block() (threadIdx.x+threadIdx.y*blockDim.x) // 2D block
#define get_tid_per_block() (threadIdx.x)

#define CHECK(res) if(res!=hipSuccess){exit(-1);}

using namespace std;
typedef float FLOAT;
#define N 20
__constant__ FLOAT A[N]; // 声明常量内存

__global__ void kernel(FLOAT *dev_b)
{
    int idx=threadIdx.x;
    if(idx>=N) return;
    dev_b[idx]=A[idx]+2;
}

int main()
{
    FLOAT *host_a=NULL;
    int nBytes=N*sizeof(FLOAT);
    HANDLE_ERROR(hipHostMalloc((void **)&host_a,nBytes));
    for(int i=0;i<N;++i) host_a[i]=1;

    // 将host 拷贝到GPU
    // hipMemcpy()  ,hipMemcpyHostToDevice默认拷贝到GPU的全局内存
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(A),host_a,nBytes));// host 到GPU常量内存

    FLOAT *dev_b=NULL,*host_b=NULL;
    HANDLE_ERROR(hipHostMalloc((void **)&host_b,nBytes));
    HANDLE_ERROR(hipMalloc((void **)&dev_b,nBytes));

    kernel<<<1,32>>>(dev_b);
    HANDLE_ERROR(hipMemcpy(host_b,dev_b,nBytes,hipMemcpyDeviceToHost));
	// hipDeviceSynchronize(); //等待GPU执行完成， 有多种方式

    // print
    for(int i=0;i<N;++i) cout<<host_b[i]<<" ";

    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    return 0;
}

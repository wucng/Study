#include "hip/hip_runtime.h"
#include "cSample.h" 
#include <hip/hip_runtime.h>

template <typename T>
__global__ void gpu_relu(int n,T* d_arr)
{
    int idx=threadIdx.x+blockDim.x*blockIdx.x;
    while(idx<n)
    {
        d_arr[idx]=d_arr[idx]<0?0:d_arr[idx];
        idx += gridDim.x * blockDim.x;
    }
}

int relu(int n,float* a_inOut)
{
    // cpu-->gpu
    float *d_inOut=NULL;
    hipMalloc((void**)&d_inOut,n*sizeof(float));
    hipMemcpy(d_inOut, a_inOut, n*sizeof(float), hipMemcpyHostToDevice);

    const int blockSize = 512;
    const int gridSize = (n + blockSize - 1) / blockSize;
    gpu_relu<float><<<gridSize,blockSize>>>(n,d_inOut);

    // GPU-->CPU
    hipMemcpy(a_inOut,d_inOut, n*sizeof(float), hipMemcpyDeviceToHost);

    // free
    hipFree(d_inOut);
    
    return 0;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>
#include <cmath>
#include <ctime>
#include "common/book.h"

#define mycout cout<<"["<<__FILE__<<":"<<__LINE__<<"] "

/* 全局线程id get thread id: 1D block and 2D grid  <<<(32,32),32>>>*/
#define get_tid() (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x)  // 2D grid,1D block
// #define get_tid() (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x+threadIdx.y*blockDim.x)  // 2D grid,2D block

/* get block id: 2D grid */
#define get_bid() (blockIdx.x + blockIdx.y * gridDim.x)

/* 每个block的线程id*/
// #define get_tid_per_block() (threadIdx.x+threadIdx.y*blockDim.x) // 2D block
#define get_tid_per_block() (threadIdx.x)

#define power(x) ((int)pow(2,x))
#define mylog2(x) ((int)log2(x))

using namespace std;
typedef float FLOAT;


__global__
void global_scan(const FLOAT *dev_x,FLOAT *dev_z,const int N,const int limit)
{
    // 由于存在跨block访问(block>1)，因此不能使用共享内存方式
    // 如果block=1，则不存在跨block访问，可以使用共享内存
    // 如果要使用共享内存，则必须消除跨block访问的问题，否则不能使用共享内存方式
    int idx=get_tid();
    if(idx>=N) return; // 越界问题处理

    dev_z[idx]=dev_x[idx];
    __syncthreads(); // 同步

    FLOAT tmp=0; // 每个线程内的局部变量

    for(int i=0;i<limit;++i)
    {
        if(idx+power(i)>=N) return; // 越界处理
        tmp=dev_z[idx]+dev_z[idx+power(i)]; // 先不写入，等待这一组完成后在统一写入，避免读写冲突问题
        __syncthreads(); // 同步

        // 写入
        dev_z[idx+power(i)]=tmp;
        __syncthreads(); // 同步
    }
}

__global__
void shared_scan(const FLOAT *dev_x,FLOAT *dev_z,const int N,const int limit)
{
    // 如果block>1,会存在跨block访问的情况，而无法使用共享内存
    // 如果block=1,不存在跨block访问的情况，可以使用共享内存

    extern __shared__ FLOAT sdatas[];// 声明共享内存
    // 由于存在跨block访问(block>1)，因此不能使用共享内存方式
    // 如果block=1，则不存在跨block访问，可以使用共享内存
    // 如果要使用共享内存，则必须消除跨block访问的问题，否则不能使用共享内存方式
    int idx = get_tid(); // 全局索引，针对全局内存访问 （以每个线程为计算单元）
    int tid = get_tid_per_block();  //每个block中threads的Idx，针对每个block的共享内存访问 （以每个block为计算单元（以每个block内所有线程组为计算单元））
    //int bid = get_bid(); // block的索引（存储每个block中间结果的访问索引）（每个block的索引）

    if(idx>=N) return; // 越界问题处理

    sdatas[tid]=dev_x[idx];
    __syncthreads(); // 同步

    FLOAT tmp=0; // 每个线程内的局部变量

    for(int i=0;i<limit;++i)
    {
        if(tid+power(i)>=N) return; // 越界处理
        tmp=sdatas[tid]+sdatas[tid+power(i)]; // 先不写入，等待这一组完成后在统一写入，避免读写冲突问题
        __syncthreads(); // 同步

        // 写入
        sdatas[tid+power(i)]=tmp;
        __syncthreads(); // 同步
    }

    // 再写入到全局变量
    dev_z[idx]=sdatas[tid];
}

int main()
{
    mycout<<"scan实现"<<endl;

    int N=1<<9; // 必须是2^n

    FLOAT *host_x=NULL,*host_z=NULL;
    FLOAT *dev_x=NULL,*dev_z=NULL;

    /**1D block*/
    int bs=1024;

    /**1D grid*/
    int grid=ceil(1.0*N/bs);

    int nbytes=N*sizeof(FLOAT);

    // 分配内存
    HANDLE_ERROR(hipHostMalloc((void **)&host_x, nbytes));
    HANDLE_ERROR(hipHostMalloc((void **)&host_z, nbytes));

    HANDLE_ERROR(hipMalloc((void **)&dev_x, nbytes));
    HANDLE_ERROR(hipMalloc((void **)&dev_z, nbytes));

    // 赋值
    for(int i=0;i<N;++i)
    {
        host_x[i]=i;
    }

    // CPU-->GPU
    HANDLE_ERROR(hipMemcpy(dev_x, host_x, nbytes, hipMemcpyHostToDevice));
    // HANDLE_ERROR(hipDeviceSynchronize()); // CPU 等待GPU操作完成


    // CPU 启动 GPU kernel计算
    {
        if (grid>1)
        {
            // 使用全局变量
            mycout<<"grid = "<<grid<<"\n使用全局内存"<<endl;
            global_scan<<<grid,bs>>>(dev_x,dev_z,N,mylog2(N));
        }
        else
        {
            // 如果blockDim.x>1 会存在跨block访问情况，共享内存没法实现block间通信(不能使用共享内存，只能使用全局内存)
            // 如果blockDim.x=1，也就是只有一个block 不存在跨block访问情况,可以使用共享内存
            mycout<<"grid = "<<grid<<"\n使用共享内存"<<endl;
            // 使用共享内存变量 (共享内存变量只能block内部通信，跨block没法访问)
            shared_scan<<<grid,bs,bs*sizeof(FLOAT)>>>(dev_x,dev_z,N,mylog2(N));
        }
    }

    // GPU-->CPU
    HANDLE_ERROR(hipMemcpy(host_z,dev_z, nbytes, hipMemcpyDeviceToHost));
    // HANDLE_ERROR(hipDeviceSynchronize());


    // 打印部分结果
    mycout<<"before scan"<<endl;
    for(int i=0;i<20;++i)
    {
        cout<<host_x[i]<<" ";
    }
    cout <<endl;

    mycout<<"after scan"<<endl;
    for(int i=0;i<20;++i)
    {
        cout<<host_z[i]<<" ";
    }
    cout <<endl;

    // 释放内存
    HANDLE_ERROR(hipFree(dev_x));
    HANDLE_ERROR(hipFree(dev_z));

    // hipHostMalloc 释放方式
    HANDLE_ERROR(hipHostFree(host_x));
    HANDLE_ERROR(hipHostFree(host_z));

    return 0;
}
